#include "hip/hip_runtime.h"
/*************************************************************************
/* GPU Programmming
/* Author: Xuehai He
/* Copyright 2020
/* University of California, San Diego
/*************************************************************************/
#include <stdio.h>

//__device__ short* c_action;
short *d_action;

__global__ void Agent_action(int2* cstate, short* next_action) {
	int i = (*cstate).y; // location
	int j = (*cstate).x;

	int r = 0;
	int d = 1;
	int l = 2;

	if (i == 0 && j == 0) {
		*next_action = r;
	}
	else if (i == 0 && j == 1) {
		*next_action = r;
	}
	else if (i == 0 && j == 2) {
		*next_action = r;
	}
	else if (i == 0 && j == 3) {
		*next_action = d;
	}
	else if (i == 1 && j == 3) {
		*next_action = d;
	}
	else if (i == 2 && j == 3) {
		*next_action = l;
	}
	else { printf("The current position is: (%d, %d)", (*cstate).x, (*cstate).y); }
}

void agent_init()
{
	// add your codes
	hipMalloc((void**)&d_action, sizeof(short));
	// *d_action = 0;
}

short* agent_action(int2* cstate)
{
	Agent_action << <1, 1 >> > (cstate, d_action);
	return d_action;
}